
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
using namespace std;

#define BUF_LEN 16
#define N 1
typedef unsigned long(*pFdummy)(void);

__device__ __noinline__ unsigned long dummy1()
{
	return 0x1111111111111111;
}
__device__ __noinline__ unsigned long dummy2()
{
	return 0x2222222222222222;
}
__device__ __noinline__ unsigned long dummy3()
{
	return 0x3333333333333333;
}
__device__ __noinline__ unsigned long dummy4()
{
	return 0x4444444444444444;
}
__device__ __noinline__ unsigned long dummy5()
{
	return 0x5555555555555555;
}
__device__ __noinline__ unsigned long dummy6()
{
	return 0x6666666666666666;
}
__device__ __noinline__ unsigned long dummy7()
{
	return 0x7777777777777777;
}
__device__ __noinline__ unsigned long dummy8()
{
	return 0x8888888888888888;
}
__device__ __noinline__ unsigned long dummy9()
{
	return 0x9999999999999999;
}

__device__ __noinline__ unsigned long unsafe(unsigned int *input,int len)
{
	unsigned int buf[BUF_LEN];
	pFdummy fp[8];
	fp[0]=dummy1;
	fp[1]=dummy2;
	fp[2]=dummy3;
	fp[3]=dummy4;
	fp[4]=dummy5;
	fp[5]=dummy6;
	fp[6]=dummy7;
	fp[7]=dummy8;
	unsigned int hash=5381;
	//copy input to buf
	

	for(int i=0;i<len;i++)
	{
		buf[i]=input[i];//通过调整len，buf可能被重写
	}


	//djb2
	for(int i=0;i<BUF_LEN;i++)
	{
		hash=((hash<<5)+hash)+buf[i];//计算input的hash值
		printf("%d\n", hash%8 );
	}
	return (unsigned long) (fp[hash%8])();//返回前面8个dummy
}

__global__ void test_kernel(unsigned long *hashes,unsigned int *input,int len,int admin)
{
	unsigned long my_hash;
	//int m;
	//m=*len;
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	printf("idx: %d, len: %d\n", idx, len);

	if(admin)//如果admin值不为0，调用dummy9，否则使用unsafe判断，调用前8个dummy
		my_hash=dummy9();
	else
		my_hash=unsafe(input+(len*idx),len);
	hashes[idx]=my_hash;
}

static void checkCudaErrorAux(const char*file,unsigned line,const char*statement,hipError_t error)
{
	if(error==hipSuccess)
		return;
	cout<<statement<<"returned:"<<hipGetErrorString(error)<<"at file:"<<file<<"line:"<<line<<endl;
	exit(1);
}
#define CUDA_CHECK_RETURN(value) checkCudaErrorAux(__FILE__,__LINE__,#value,value)

int main()
{
	unsigned int input[100];
	int len=27,admin=0;
	unsigned long hashes[N];
	unsigned long *dev_hashes;
	unsigned int *dev_input;
	unsigned int m=0;
	
	m=0x250;

	//cout<<"start!"<<endl;
		for(int i=0;i<len;i++)
			input[i]=m;

	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_hashes,N*sizeof(unsigned long)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_input,100*sizeof(unsigned int)));
	CUDA_CHECK_RETURN(hipMemcpy(dev_input,input,100*sizeof(unsigned int),hipMemcpyHostToDevice));

	test_kernel<<<1,1>>>(dev_hashes,dev_input,len,admin);
	CUDA_CHECK_RETURN(hipMemcpy(hashes,dev_hashes,N*sizeof(unsigned long),hipMemcpyDeviceToHost));

	for(int i=0;i<N;i++)
	{
		printf("%lx\n", hashes[i]);
	}

	CUDA_CHECK_RETURN(hipFree(dev_input));
	CUDA_CHECK_RETURN(hipFree(dev_hashes));
	
	return 0;
}
